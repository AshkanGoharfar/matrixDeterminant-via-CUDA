
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

const int BLOCKS_PER_GRID = 5;
const int THREADS_PER_BLOCK = 20;
const int N=5;

void printMatrix(int* X){
	printf("%3d\n",X[0]);
	for(int i=0;i<X[0];i++){
		for(int j=0;j<X[0];j++){
			printf("%4d",X[i*X[0]+j+1]);
		}
		printf("\n");
	}
	printf("\n");
}

__global__
void det(int* M,int* w,int* tab){
	int N = M[0];
	int idx = threadIdx.x + blockDim.x * blockIdx.x;	
	int stride = blockDim.x * gridDim.x;

	//const int shared = blockDim.x;
        
	//__shared__ int sharedTab[THREADS_PER_BLOCK];

	int* idxTab = new int[N];
	for(int i=1;i<=N;i++){
		idxTab[N-i] = idx%i;
		idx/=i;
	}
	//idxTab[0]+=idx;

	if(idx==0){
		int* strideTab = new int[N];
		for(int i=1;i<=N;i++){
			strideTab[N-i] = stride%i;
			stride/=i;
		}
		//strideTab[0]+=stride;

		idx = threadIdx.x + blockDim.x * blockIdx.x;	
		stride = blockDim.x * gridDim.x;
		
		/*__syncthreads();
		//printf("N: %3d,idx: %3d,stride: %3d",N,idx,stride);
		for(int i=0;i<N;i++){
			printf("N: %3d,idx: %3d,stride: %3d,idxTab[%d]==%d\n",N,idx,stride,i,idxTab[i]);
		}
		for(int i=0;i<N;i++){
			printf("N: %3d,idx: %3d,stride: %3d,stride[%d]==%d\n",N,idx,stride,i,strideTab[i]);
		}*/
		while(idxTab[0]<N){	
			//int parz = (idxTab[N-3]+idxTab[N-2]+idxTab[N-1])%2; //0 lub 1(wskaznik parzystosci)
			int parz = 0;
			for(int i=0;i<N;i++) parz = (parz + idxTab[i])%2;
			//conv
			for(int i=0;i<N;i++){
				for(int j=1;j<=N;j++){
					bool niepojawilo = true;
					for(int k=0;k<i;k++){
						if(idxTab[k] == j){niepojawilo = false; break;}
					}
					if(niepojawilo){
						if(idxTab[i]==0){
							idxTab[i]=j;break;
						}
						else idxTab[i]--;
					}
				}
			}

			//idxTab zawiera teraz interesuja permutacje
			int product = ((parz%2==0) ? 1 : -1);//trzeba bedzie jakis inny typ, pewnie double albo klase, bo to duzo wychodzi i w incie sie nie miesci
			for(int i=0;i<N;i++) product*=M[i*N+(idxTab[i]-1)+1]; //here we have a product, one of N!
		
			//__syncthreads();
			tab[idx] += product;
			
			//sharedTab[idx%stride] = product;

			/*if(idx%THREADS_PER_BLOCK==0){
				int nr_bloku = blockIdx.x;
				for(int i=0;i<THREADS_PER_BLOCK;i++){
					tab[nr_bloku] += sharedTab[i]; 
				}
				printf("nr_bloku: %d, tab[i]=%d",nr_bloku,tab[nr_bloku]);
			}*/

				
			/*if(idx==0){
				for(int i=0;i<stride;i++) (*w)+=tab[i];
			}*/
			//__syncthreads();
			
			//(*w)+=product;

			//printf("Idx: %2d, product: %3d,tab[%d]: %d\n",idx,product,idx,tab[idx]);
			
			//negconv
			for(int i=0;i<N;i++){
				int ile = 0;
				for(int j=i+1;j<N;j++){
					if(idxTab[j]<idxTab[i]) ile++;
				}
				idxTab[i] = ile;
			}
			
			//idxTab+=strideTab
			int ak=0;
			for(int i=1;i<=N;i++){
				idxTab[N-i]=idxTab[N-i]+strideTab[N-i]+ak;
				ak=idxTab[N-i]/i;
				if(i!=N) idxTab[N-i]%=i;		
			}
			idxTab[0]+=ak;
			/*ak=0;
			for(int i=1;i<=N;i++){
				idxTab[N-i]=idxTab[N-i]+strideTab[N-i]+ak;
				ak=idxTab[N-i]/i;
				if(i!=N) idxTab[N-i]%=i;		
			}
			idxTab[0]+=ak;
			ak=0;
			for(int i=1;i<=N;i++){
				idxTab[N-i]=idxTab[N-i]+strideTab[N-i]+ak;
				ak=idxTab[N-i]/i;
				if(i!=N) idxTab[N-i]%=i;		
			}
			idxTab[0]+=ak;*/

			}
		delete[] strideTab;
	}
	/*__syncthreads();
		//printf("N: %3d,idx: %3d,stride: %3d",N,idx,stride);
		for(int i=0;i<N;i++){
			printf("N: %3d,idx: %3d,stride: %3d,idxTab[%d]==%d\n",N,idx,stride,i,idxTab[i]);
		}
		for(int i=0;i<N;i++){
			printf("N: %3d,idx: %3d,stride: %3d,stride[%d]==%d\n",N,idx,stride,i,strideTab[i]);
		}*/
	delete[] idxTab;

	/*for(int i=0;i<BLOCKS_PER_GRID*THREADS_PER_BLOCK;i++){
		(*w) += tab[i];
	}*/
}

int main(){
	srand(time(NULL));
	//const int N = 5;
	int* tab;// = new int[BLOCKS_PER_GRID*THREADS_PER_BLOCK];
	hipMallocManaged(&tab, sizeof(int)*BLOCKS_PER_GRID*THREADS_PER_BLOCK);
	for(int i=0;i<BLOCKS_PER_GRID*THREADS_PER_BLOCK;i++) tab[i] = 0;
	
	int *A;

	hipMallocManaged(&A, (1+N*N)*sizeof(int));

	A[0] = N;

	for(int i=0;i<A[0];i++){
		for(int j=0;j<A[0];j++){
			A[i*A[0]+j+1] = ((i==j) ? 3.0 : 2.0);//rand()%21-10;
		}
	}

	printMatrix(A);

	clock_t start = clock();

	int* w = new int; *w = 0;
	det<<<BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(A,w,tab);
	hipDeviceSynchronize();

	for(int i=0;i<BLOCKS_PER_GRID*THREADS_PER_BLOCK;i++) (*w)+=tab[i];	
	printf("%5d\n",*w);

	hipFree(A);	
	hipFree(tab);
	delete w;

	clock_t koniec = clock();
	double czas = (double)(koniec-start)/CLOCKS_PER_SEC;
	printf("Czas wykonania: %lfs\n",czas);

	return 0;
}
